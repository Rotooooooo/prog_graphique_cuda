#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>

 __global__ void grayscale_laplacian_shared( unsigned char * rgb, unsigned char * tab_res, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * (blockDim.x-5) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-5) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  auto w = blockDim.x;
  auto h = blockDim.y;

  extern __shared__ unsigned char shared_tab[];

  if( i < cols && j < rows ) {
    shared_tab[ lj * w + li ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) >> 10;
  }

  __syncthreads();

  if( i < cols -2 && j < rows-2 && li > 2 && li < (w-2) && lj > 2 && lj < (h-2) )
  {
    auto res = 
           -     shared_tab[ (lj-1)*w + li - 1 ] - 2* shared_tab[ (lj-1)*w + li ] -   shared_tab[ (lj-1)*w + li + 1 ]
           - 2 * shared_tab[ (lj  )*w + li - 1 ] + 16*shared_tab[ (lj  )*w + li ] - 2*shared_tab[ (lj  )*w + li + 1 ]
           -     shared_tab[ (lj+1)*w + li - 1 ] - 2* shared_tab[ (lj+1)*w + li ] -   shared_tab[ (lj+1)*w + li + 1 ]
           -     shared_tab[ (lj-2)*w + li     ] -    shared_tab[ (lj  )*w + li - 2 ] - shared_tab[ (lj+2)*w + li] - shared_tab[ (lj)*w + li + 2 ];

    res = res > 255 ? 255 : res;
    res = res < 0 ? 0 : res;
    tab_res[ j * cols + i ] = res;
  }
}


int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );

  auto rows = m_in.rows;
  auto cols = m_in.cols;
 
  unsigned char * rgb = nullptr;
  hipHostMalloc( &rgb, 3 * rows * cols );

  std::memcpy( rgb, m_in.data, 3 * rows * cols );

  unsigned char * rgb_d;
  unsigned char * tab_d;
  unsigned char * s_d;

  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &tab_d, rows * cols );
  hipMalloc( &s_d, rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );

  dim3 block( 64, 8 );
  dim3 grid1( ( cols - 1) / (block.x-5) + 1 , ( rows - 1 ) / (block.y-5) + 1 );

  hipStream_t stream[2];
  hipStreamCreate(&stream[0]);
  hipStreamCreate(&stream[1]);

  hipEvent_t start, stop;

  hipEventCreate( &start );
  hipEventCreate( &stop );

  // Mesure du temps de calcul du kernel
  hipEventRecord( start );

  grayscale_laplacian_shared<<< grid1, block, block.x * (block.y+2)*sizeof(unsigned char),stream[0] >>>( rgb_d, s_d, cols, rows/2 + 2);
  grayscale_laplacian_shared<<< grid1, block, block.x * (block.y+2)*sizeof(unsigned char),stream[1] >>>(rgb_d+(((rows*cols*3)/2)-cols*3*3),tab_d,cols,rows/2 +3);

  unsigned char* tab = nullptr;
  hipHostMalloc(&tab,rows*cols);
  hipMemcpyAsync(tab,s_d,(rows*cols)/2,hipMemcpyDeviceToHost,stream[0]);
  hipMemcpyAsync(tab+(rows*cols)/2,tab_d+cols*3,(rows*cols)/2,hipMemcpyDeviceToHost,stream[1]);


  cv :: Mat m_out(rows,cols,CV_8UC1,tab);

  hipEventRecord( stop );

  hipEventSynchronize( stop );
  float duration;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "time=" << duration << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  cv::imwrite( "out.jpg", m_out );

  hipFree( rgb_d);
  hipFree( tab_d);
  hipFree( s_d);

  hipHostFree( tab );
  hipHostFree( rgb );
  hipStreamDestroy(stream[0]);
  hipStreamDestroy(stream[1]);

  return 0;
}

