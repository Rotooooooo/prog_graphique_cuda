#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>

/**
 * Kernel pour transformer l'image RGB en niveaux de gris.
 */
__global__ void grayscale( unsigned char * rgb, unsigned char * tab, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < cols && j < rows ) {
    tab[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) >> 10;
  }
}

/**
 * Kernel pour flouter l'image à partir de l'image en niveaux de gris.
 */
__global__ void blur( unsigned char * tab, unsigned char * res, std::size_t cols, std::size_t rows )
{
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;

  if( i > 1 && i < cols && j < rows && j > 1)
  {
    auto somme_pixel =
           tab[ (j-1)*cols + i - 1 ] + tab[ (j-1)*cols + i] + tab[ (j-1)*cols + i + 1 ] +
           tab[ (j  )*cols + i - 1 ] + tab[ (j  )*cols + i] + tab[ (j  )*cols + i + 1 ] +
           tab[ (j+1)*cols + i - 1 ] + tab[ (j+1)*cols + i] + tab[ (j+1)*cols + i + 1 ];


    auto moyenne_pixel = somme_pixel/9;
    moyenne_pixel = moyenne_pixel > 255 ? moyenne_pixel = 255 : moyenne_pixel;

    res[ j * cols + i ] = moyenne_pixel;
  }
}


/**
 * Kernel pour flouter l'image à partir de l'image en niveaux de gris, en utilisant la mémoire shared
 * pour limiter les accès à la mémoire globale.
 */
__global__ void blur_shared( unsigned char * tab, unsigned char * res, std::size_t cols, std::size_t rows )
{
  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  auto w = blockDim.x;
  auto h = blockDim.y;

  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  extern __shared__ unsigned char shared_tab[];

  if( i < cols && j < rows )
  {
    shared_tab[ lj * w + li ] = tab[ j * cols + i ];
  }

  __syncthreads();

  if( i < cols -1 && j < rows-1 && li > 0 && li < (w-1) && lj > 0 && lj < (h-1) )
  {
    auto somme_pixel =
      shared_tab[ (lj-1)*w + li - 1 ] + shared_tab[ (lj-1)*w + li] + shared_tab[ (lj-1)*w + li + 1 ] +
      shared_tab[ (lj  )*w + li - 1 ] + shared_tab[ (lj  )*w + li] + shared_tab[ (lj  )*w + li + 1 ] +
      shared_tab[ (lj+1)*w + li - 1 ] + shared_tab[ (lj+1)*w + li] + shared_tab[ (lj+1)*w + li + 1 ];


    auto moyenne_pixel = somme_pixel/9;
    moyenne_pixel = moyenne_pixel > 255 ? moyenne_pixel = 255 : moyenne_pixel;

    res[ j * cols + i ] = moyenne_pixel;
  }
}


/**
 * Kernel fusionnant le passage en niveaux de gris et le flou.
 */
__global__ void grayscale_blur_shared( unsigned char * rgb, unsigned char * s, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  auto w = blockDim.x;
  auto h = blockDim.y;

  extern __shared__ unsigned char shared_tab[];

  if( i < cols && j < rows ) {
    shared_tab[ lj * w + li ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) >> 10;
  }

  /**
   * Il faut synchroniser tous les warps (threads) du bloc pour être certain que le niveau de gris est calculé
   * par tous les threads du bloc avant de pouvoir accéder aux données des pixels voisins.
   */
  __syncthreads();

  if( i < cols -1 && j < rows-1 && li > 0 && li < (w-1) && lj > 0 && lj < (h-1) )
  {
    auto somme_pixel =
      shared_tab[ (lj-1)*w + li - 1 ] + shared_tab[ (lj-1)*w + li] + shared_tab[ (lj-1)*w + li + 1 ] +
      shared_tab[ (lj  )*w + li - 1 ] + shared_tab[ (lj  )*w + li] + shared_tab[ (lj  )*w + li + 1 ] +
      shared_tab[ (lj+1)*w + li - 1 ] + shared_tab[ (lj+1)*w + li] + shared_tab[ (lj+1)*w + li + 1 ];


    auto moyenne_pixel = somme_pixel/9;
    moyenne_pixel = moyenne_pixel > 255 ? moyenne_pixel = 255 : moyenne_pixel;

    s[ j * cols + i ] = moyenne_pixel;
  }
}


int main()
{
  cv::Mat m_in = cv::imread("casquette.jpg", cv::IMREAD_UNCHANGED );

  auto rows = m_in.rows;
  auto cols = m_in.cols;


  unsigned char * tab = nullptr;
  hipHostMalloc( &tab, rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, tab );

 
  unsigned char * rgb = nullptr;
  hipHostMalloc( &rgb, 3 * rows * cols );

  std::memcpy( rgb, m_in.data, 3 * rows * cols );

  unsigned char * rgb_d;
  unsigned char * tab_d;
  unsigned char * s_d;

  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &tab_d, rows * cols );
  hipMalloc( &s_d, rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );

  dim3 block( 32, 4 );
  dim3 grid0( ( cols - 1) / block.x + 1 , ( rows - 1 ) / block.y + 1 );

  dim3 grid1( ( cols - 1) / (block.x-2) + 1 , ( rows - 1 ) / (block.y-2) + 1 );

  hipEvent_t start, stop;

  hipEventCreate( &start );
  hipEventCreate( &stop );

  // Mesure du temps de calcul du kernel
  hipEventRecord( start );

  /*
  // Version en 2 étapes.
  grayscale<<< grid0, block >>>( rgb_d, tab_d, cols, rows );
  blur<<< grid0, block >>>( tab_d, s_d, cols, rows );
  */

  // Version en 2 étapes, blur avec mémoire shared.
  grayscale<<< grid0, block >>>( rgb_d, tab_d, cols, rows );
  blur_shared<<< grid1, block, block.x * block.y >>>( tab_d, s_d, cols, rows );
  

  // Version fusionnée.
  //grayscale_blur_shared<<< grid1, block, block.x * block.y >>>( rgb_d, s_d, cols, rows );

  hipEventRecord( stop );

  hipMemcpy( tab, s_d, rows * cols, hipMemcpyDeviceToHost );

  hipEventSynchronize( stop );
  float duration;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "time=" << duration << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  cv::imwrite( "casquette_box_blur.jpg", m_out );

  hipFree( rgb_d);
  hipFree( tab_d);
  hipFree( s_d);

  hipHostFree( tab );
  hipHostFree( rgb );

  return 0;
}

